
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include<math.h>
#include <fstream>
#include <Windows.h>
#include "RSA.h"
double performancecounter_diff(LARGE_INTEGER *a, LARGE_INTEGER *b)
{
	LARGE_INTEGER freq;
	QueryPerformanceFrequency(&freq);
	return (double)(a->QuadPart - b->QuadPart) / (double)freq.QuadPart;
}
int main()
{

	RSA a(1024);
	string s = a.cifrarMensaje("Mi mama Me Mi r 0876543 LOL");
	cout << "Mensaje Encriptado: " << s;
	cout << a.descifrarMensaje(s) << endl;
	system("Pause");
	return 0;
}
