#include "hip/hip_runtime.h"
#include "aritmetica.h"
#include "aritmetica.h"

__global__
void FFTKernel(double* data, unsigned long istep, unsigned long m, unsigned long mmax, double wr, double wi, unsigned long tam){
	int i = (threadIdx.x + blockDim.x * blockIdx.x)*(istep)+(m);
	if (i <= (tam))
	{
		unsigned long j = i + (mmax);
		double tempr = (wr)* data[j - 1] - (wi)* data[j];
		double tempi = (wr)* data[j] + (wi)* data[j - 1];

		data[j - 1] = (data[i - 1] - tempr);
		data[j] = (data[i] - tempi);
		data[i - 1] = (data[i - 1] + tempr);
		data[i] = (data[i] + tempi);
	}
}
void fourpara(double* data, unsigned long* istep, unsigned long* m, unsigned long* mmax, double* wr, double* wi, unsigned long* tam)
{
	int size = (*tam) * sizeof(double);
	double *d_data;
	hipMalloc((void **)&d_data, size);
	hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
	FFTKernel << < ceil(*tam/1024), 1024 >> > (d_data, *istep, *m, *mmax, *wr, *wi, *tam);
	hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
	hipFree(d_data);
}
void aritmetica::four(double* data, unsigned long nn)
{
	unsigned long n, mmax, m, j, istep, i;
	double wtemp, wr, wpr, wpi, wi, theta;

	n = nn << 1;
	j = 1;
	for (i = 1; i<n; i += 2) {
		if (j>i) {
			swap(data[j - 1], data[i - 1]);
			swap(data[j], data[i]);
		}
		m = nn;
		while (m >= 2 && j>m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	};
	mmax = 2;
	while (n>mmax) {
		istep = mmax << 1;
		theta = -(2 * M_PI / mmax);
		wtemp = sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi = sin(theta);
		wr = 1.0;
		wi = 0.0;
		for (m = 1; m < mmax; m += 2) {
			fourpara(data, &istep, &m, &mmax, &wr, &wi, &n);
			wtemp = wr;
			wr += wr*wpr - wi*wpi;
			wi += wi*wpr + wtemp*wpi;
		}
		mmax = istep;
	}
}

__global__
void IFFTKernel(double* data, unsigned long istep, unsigned long m, unsigned long mmax, double wr, double wi, unsigned long tam){
	int i = (threadIdx.x + blockDim.x * blockIdx.x)*(istep)+(m);
	if (i <= (tam))
	{
		unsigned long j = i + (mmax);
		double tempr = (wr)* data[j - 1] - (wi)* data[j];
		double tempi = (wr)* data[j] + (wi)* data[j - 1];

		data[j - 1] = (data[i - 1] - tempr) / 2;
		data[j] = (data[i] - tempi) / 2;
		data[i - 1] = (data[i - 1] + tempr) / 2;
		data[i] = (data[i] + tempi) / 2;
	}
}
void fourIpara(double* data, unsigned long* istep, unsigned long* m, unsigned long* mmax, double* wr, double* wi, unsigned long* tam)
{
	int size = (*tam) * sizeof(double);
	double *d_data;
	hipMalloc((void **)&d_data, size);
	hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
	IFFTKernel << <ceil(*tam / 1024), 1024 >> > (d_data, *istep, *m, *mmax, *wr, *wi, *tam);
	hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost);
	hipFree(d_data);
}
void aritmetica::fourI(double* data, unsigned long nn)
{
	unsigned long n, mmax, m, j, istep, i;
	double wtemp, wr, wpr, wpi, wi, theta;

	n = nn << 1;
	j = 1;
	for (i = 1; i<n; i += 2) {
		if (j>i) {
			swap(data[j - 1], data[i - 1]);
			swap(data[j], data[i]);
		}
		m = nn;
		while (m >= 2 && j>m) {
			j -= m;
			m >>= 1;
		}
		j += m;
	};
	mmax = 2;
	while (n>mmax) {
		istep = mmax << 1;
		theta = -(2 * M_PI / mmax);
		wtemp = sin(0.5*theta);
		wpr = -2.0*wtemp*wtemp;
		wpi = sin(theta);
		wr = 1.0;
		wi = 0.0;
		for (m = 1; m < mmax; m += 2) {
			fourIpara(data, &istep, &m, &mmax, &wr, &wi, &n);
			wtemp = wr;
			wr += wr*wpr - wi*wpi;
			wi += wi*wpr + wtemp*wpi;
		}
		mmax = istep;
	}
}
void aritmetica::MultComple(double *X, double *Xi, double *Y, double *Yi, double* Resp, double* Respi)
{
	if (*Xi == 0 && *Yi == 0)
		*Resp = (*X) * (*Y);
	else
	{
		*Resp = ((*X) * (*Y)) - ((*Xi) * (*Yi));
		*Respi = ((*X) * (*Yi)) + ((*Xi) * (*Y));
	}
}
void aritmetica::ConjuComple(double* Resp, int tam)
{
	for (int i = 1; i < tam; i += 2)
	{
		if (Resp[i] != 0)
			Resp[i] = -Resp[i];
	}
}
double* aritmetica::Mult(double* X, double* Y, unsigned long tam)
{
	double* Resp = new double[tam];
	memset(Resp, 0, tam*sizeof(double));
	four(X, tam / 2);
	four(Y, tam / 2);
	for (unsigned long i = 0; i < tam; i += 2)
	{
		MultComple(X + i, X + i + 1, Y + i, Y + i + 1, Resp + i, Resp + i + 1);
	}
	ConjuComple(Resp, tam);
	fourI(Resp, tam / 2);
	return Resp;
}
ZZ aritmetica::powM(ZZ a, ZZ m, ZZ modulo,int NumB)
{

	ZZ respuesta;
	respuesta = 1;
	ZZ x;
	x = a;
	while (m != 0)
	{

		if ((m & 1) == 1)
		{
			respuesta = (MultiFourier(respuesta,x,NumB)) % modulo;
			//cout<<"respuesta_ "<<respuesta<<" x: "<<x<<endl;


		}
		x = (MultiFourier(x, x, NumB)) % modulo;
		m >>= 1;
		// cout<<"m: "<<m<<" x: "<<x<<" respuesta: "<<respuesta<<endl;
		//if(mod(x,modulo)==1) break;

	}
	//cout<<endl;
	return respuesta;
}
ZZ aritmetica::Blum(long n)
{
    ZZ N,semilla, p, q, bits,x, res, temp;

    p = to_ZZ("7171153257");
    q =to_ZZ("5");

    N = p * q;
    clock_t t;
    t=clock();
    semilla =t;
    x = semilla%N;
    res = 0, bits = 0;

    #pragma omp parallel for
    for(int i=n; i>0; i--)
    {
        x = powM(x, to_ZZ(2), N, n);
        bits = x-((x>>1)<<1);
        power(temp,to_ZZ(2),(i-1));
        res += bits*temp;
    }
    return res;
}
ZZ aritmetica::aleatorioBits(long long i)
{
    ZZ d =Blum(i);
    // cout<<d<<endl;
    if((d &1)==0)
        return d+1;
    else
        return d;
}
/*ZZ aritmetica::generaPrimo(long long bits)
{
    ZZ n =aleatorioBits(bits);
    while(MillerWitness((n),to_ZZ(80))==0)
        n=aleatorioBits(bits);
    return n;
}*/
ZZ aritmetica::generaPrimo(long long bits)
{
    ZZ n;
   
   do {
        n=aleatorioBits(bits);
        //cout<<n<<endl<<endl;
        //cout<<ProbPrime(n)<<endl<<endl;
    } while(ProbPrime(n)==0);
    return n;
}
ZZ aritmetica::MultiFourier(ZZ X, ZZ Y, int NumbitsRSA)
{
	stringstream convertX;
	convertX << X;
	string SX = convertX.str();
	stringstream convertY;
	convertY << Y;
	string SY = convertY.str();
	int xt = SX.size() - 1;
	int yt = SY.size() - 1;
	double * XVec = new double[NumbitsRSA * 2];
	double * YVec = new double[NumbitsRSA * 2];
	memset(XVec, 0, NumbitsRSA * 2 * sizeof(double));
	memset(YVec, 0, NumbitsRSA * 2 * sizeof(double));
	for (int i = xt, j = 0; i >= 0; i--, j += 2)
	{
		char nume = SX.at(i);
		XVec[j] = atof(&nume);
	}

	for (int i = yt, j = 0; i >= 0; i--, j += 2)
	{
		char nume = SY.at(i);
		YVec[j] = atof(&nume);
	}
	SX.clear();
	SY.clear();
	double* Mu = Mult(XVec, YVec, NumbitsRSA * 2);
	delete(XVec);
	delete(YVec);
	ZZ Respuesta = to_ZZ(0);
	ZZ diez = to_ZZ(1);
	for (int i = 0; Mu[i] >= 1; i += 2)
	{
		string X = to_string(Mu[i]);
		Respuesta += to_ZZ(X.c_str()) * diez;
		diez *= to_ZZ(10);
	}
	return Respuesta;
}




